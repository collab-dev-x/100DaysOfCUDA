#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel to separate RGB channels
__global__ void split_channels(unsigned char* img, unsigned char* r, unsigned char* g, unsigned char* b, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        int pixelIdx = y * width + x;
        r[pixelIdx] = img[idx];
        g[pixelIdx] = img[idx + 1];
        b[pixelIdx] = img[idx + 2];
    }
}

// CUDA kernel to merge RGB channels
__global__ void merge_channels(unsigned char* img, unsigned char* r, unsigned char* g, unsigned char* b, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        int pixelIdx = y * width + x;
        img[idx]     = r[pixelIdx];
        img[idx + 1] = g[pixelIdx];
        img[idx + 2] = b[pixelIdx];
    }
}

int main() {
    int width, height, channels;
    const char* input_path = "image.jpg";
    const char* output_path = "output.png";

    // Load image (CPU)
    unsigned char* input_img = stbi_load(input_path, &width, &height, &channels, 3);
    if (!input_img) {
        std::cerr << "Failed to load image!\n";
        return -1;
    }

    size_t img_size = width * height * 3 * sizeof(unsigned char);
    size_t channel_size = width * height * sizeof(unsigned char);

    // Allocate device memory
    unsigned char *d_img, *d_r, *d_g, *d_b;
    hipMalloc(&d_img, img_size);
    hipMalloc(&d_r, channel_size);
    hipMalloc(&d_g, channel_size);
    hipMalloc(&d_b, channel_size);

    // Copy image to device
    hipMemcpy(d_img, input_img, img_size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + 15) / 16, (height + 15) / 16);
    split_channels<<<blocksPerGrid, threadsPerBlock>>>(d_img, d_r, d_g, d_b, width, height);
    hipDeviceSynchronize();

    // Merge back
    merge_channels<<<blocksPerGrid, threadsPerBlock>>>(d_img, d_r, d_g, d_b, width, height);
    hipDeviceSynchronize();

    // Copy back to host
    unsigned char* output_img = new unsigned char[width * height * 3];
    hipMemcpy(output_img, d_img, img_size, hipMemcpyDeviceToHost);

    // Save output image
    stbi_write_png(output_path, width, height, 3, output_img, width * 3);

    // Free memory
    stbi_image_free(input_img);
    delete[] output_img;
    hipFree(d_img);
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);

    std::cout << "Image processing complete. Output saved as " << output_path << "\n";
    return 0;
}
