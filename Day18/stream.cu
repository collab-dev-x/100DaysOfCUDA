#include <hip/hip_runtime.h>
#include <iostream>

#define N (1 << 24)
#define BLOCK_SIZE 256

__global__ void vectorAdd(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

void check(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A1, *d_B1, *d_C1;
    float *d_A2, *d_B2, *d_C2;

    size_t size = N * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f; h_B[i] = 2.0f;
    }

    hipMalloc(&d_A1, size); hipMalloc(&d_B1, size); hipMalloc(&d_C1, size);
    hipMalloc(&d_A2, size); hipMalloc(&d_B2, size); hipMalloc(&d_C2, size);

    // Copy input to both sets
    hipMemcpy(d_A1, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B1, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_A2, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B2, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float elapsed = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // --- Sequential
    hipEventRecord(start);
    vectorAdd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_A1, d_B1, d_C1, N);
    vectorAdd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_A2, d_B2, d_C2, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << "Sequential execution: " << elapsed << " ms\n";

    // --- Concurrent with Streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEventRecord(start);
    vectorAdd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream1>>>(d_A1, d_B1, d_C1, N);
    vectorAdd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream2>>>(d_A2, d_B2, d_C2, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << "Concurrent execution (streams): " << elapsed << " ms\n";

    // Cleanup
    hipFree(d_A1); hipFree(d_B1); hipFree(d_C1);
    hipFree(d_A2); hipFree(d_B2); hipFree(d_C2);
    free(h_A); free(h_B); free(h_C);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
