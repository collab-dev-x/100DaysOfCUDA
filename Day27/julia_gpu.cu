#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>
#include <complex>
#include <vector>

#define WIDTH 4000
#define HEIGHT 4000
#define MAX_ITER 1000
#define SCALE 1.5f

struct hipComplex {
    float r; 
    float i; 

    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __device__ float magnitude2() {
        return r * r + i * i;
    }

    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y) {
    hipComplex c(-0.8f, 0.156f);
    float jx = SCALE * (WIDTH / 2.0f - x) / (WIDTH / 2.0f);
    float jy = SCALE * (HEIGHT / 2.0f - y) / (HEIGHT / 2.0f);
    hipComplex z(jx, jy);

    for (int i = 0; i < MAX_ITER; i++) {
        z = z * z + c;
        if (z.magnitude2() > 1000.0f)
            return i;
    }
    return MAX_ITER;
}

__global__ void generate_image(unsigned char * image){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(row>=HEIGHT || col >= WIDTH)
        return;

    int iter = julia(col, row);
    int pixelIdx = 3 * ( row* WIDTH + col);
    unsigned char color = (unsigned char)(255 * iter / MAX_ITER);
    image[ pixelIdx + 0] = color;
    image[ pixelIdx + 1] = color;
    image[ pixelIdx + 2] = color; 
}

int main() {

    unsigned char *d_image, *image =  new unsigned char [WIDTH * HEIGHT * 3];
    hipMalloc(&d_image,WIDTH*HEIGHT*3*sizeof(unsigned char));

    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

    generate_image<<<gridSize,blockSize>>>(d_image);
    hipDeviceSynchronize();

    hipMemcpy(image,d_image,WIDTH*HEIGHT*3*sizeof(unsigned char),hipMemcpyDeviceToHost);

    if (stbi_write_png("julia.png", WIDTH, HEIGHT, 3, image, WIDTH * 3))
        std::cout << "Saved 'julia.png'\n";
    else
        std::cerr << "Failed to save image.\n";

    return 0;
}
