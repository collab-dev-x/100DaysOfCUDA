#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// y = αAx + β*y
__global__ void matVecMulKernel(const float* A, const float* x, float* y, float alpha, float beta, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m) {
        float dot = 0.0f;
        for (int col = 0; col < n; ++col) {
            dot += A[row * n + col] * x[col]; 
        }
        y[row] = alpha * dot + beta * y[row];
    }

}

int main() {
    const int m = 10;
    const int n = 8;

    const int num_mat = m * n;
    float h_a[num_mat];

    for(int i=0;i<num_mat;i++){
        h_a[i]=static_cast<float>(i%100);
        std::cout<<h_a[i]<<" ";
    }
    std::cout<<"\n";

    float h_x[n];
    for(int i=0;i<n;i++){
        h_x[i]=1.0f;
    }
    float h_y[m];
    for(int i=0;i<m;i++){
        h_y[i]=1.0f;
    }

    float h_y_result[m];
    float alpha = 2.0f;
    float beta = 0.6f;

    float *d_a, *d_x, *d_y;
    hipMalloc(&d_a, m * n * sizeof(float));
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, m * sizeof(float));

    hipMemcpy(d_a, h_a, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, m * sizeof(float), hipMemcpyHostToDevice);

    float kernalTime, cuBLAS_Time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int threadsPerBlock = 256;
    int blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    hipblasSgemv(handle, HIPBLAS_OP_T, n, m, &alpha, d_a, m, d_x, 1, &beta, d_y, 1);
    hipMemcpy(h_y_result, d_y, m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuBLAS_Time, start, stop);

    std::cout<<"cuBLAS Execution Time: "<<cuBLAS_Time<<std::endl;
    std::cout<<"cuBLAS Result: "<<std::endl;
    for(int i=0;i<5;i++){
        std::cout<<h_y_result[i]<<" ";
    }
    std::cout<<"\n";

    hipMemcpy(d_y, h_y, m * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(start);
    matVecMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_x, d_y, alpha, beta, m, n);
    hipMemcpy(h_y_result, d_y, m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernalTime, start, stop);

    std::cout<<"Kernal Execution Time: "<<kernalTime<<std::endl;
    std::cout<<"Kernal Result: "<<std::endl;
    for(int i=0;i<5;i++){
        std::cout<<h_y_result[i]<<" ";
    }
    std::cout<<"\n";


    


    hipFree(d_a);
    hipFree(d_x);
    hipFree(d_y);
    delete [] h_a;
    delete [] h_x;
    delete [] h_y;

    return 0;
}