#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start1, stop1, start2, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    const int M = 256, N = 256, K = 256;
    const int batch_size = 8;
    const float alpha = 1.0f, beta = 0.5f;
    
    size_t matrix_size = M * N * sizeof(float);
    size_t batch_matrix_size = batch_size * M * N * sizeof(float);
    
    float *d_A, *d_B, *d_C;
    float *d_batch_A, *d_batch_B, *d_batch_C;
    float **d_A_array, **d_B_array, **d_C_array;
    
    hipMalloc(&d_A, matrix_size);
    hipMalloc(&d_B, matrix_size);
    hipMalloc(&d_C, matrix_size);
    hipMalloc(&d_batch_A, batch_matrix_size);
    hipMalloc(&d_batch_B, batch_matrix_size);
    hipMalloc(&d_batch_C, batch_matrix_size);
    hipMalloc(&d_A_array, batch_size * sizeof(float*));
    hipMalloc(&d_B_array, batch_size * sizeof(float*));
    hipMalloc(&d_C_array, batch_size * sizeof(float*));
    
    std::vector<float> h_data(M * N, 1.0f);
    hipMemcpy(d_A, h_data.data(), matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_data.data(), matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_data.data(), matrix_size, hipMemcpyHostToDevice);
    
    for(int i = 0; i < batch_size; i++) {
        hipMemcpy(d_batch_A + i * M * N, h_data.data(), matrix_size, hipMemcpyHostToDevice);
        hipMemcpy(d_batch_B + i * M * N, h_data.data(), matrix_size, hipMemcpyHostToDevice);
        hipMemcpy(d_batch_C + i * M * N, h_data.data(), matrix_size, hipMemcpyHostToDevice);
    }
    
    std::vector<float*> h_A_array(batch_size), h_B_array(batch_size), h_C_array(batch_size);
    for(int i = 0; i < batch_size; i++) {
        h_A_array[i] = d_batch_A + i * M * N;
        h_B_array[i] = d_batch_B + i * M * N;
        h_C_array[i] = d_batch_C + i * M * N;
    }
    hipMemcpy(d_A_array, h_A_array.data(), batch_size * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_B_array, h_B_array.data(), batch_size * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_C_array, h_C_array.data(), batch_size * sizeof(float*), hipMemcpyHostToDevice);
    
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M); //C = α × A × B + β × C

    float cuBLAS_Time1, cuBLAS_Time2;

    hipEventRecord(start1);
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, 
                       (const float**)d_A_array, M, (const float**)d_B_array, K, &beta, d_C_array, M, batch_size);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&cuBLAS_Time1, start1, stop1);
    hipEventRecord(start2);
    hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, 
                              d_batch_A, M, M*N, d_batch_B, K, M*N, &beta, d_batch_C, M, M*N, batch_size);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&cuBLAS_Time2, start2, stop2);
    
    std::cout<<"CUBLAS Batched Execution Time: "<<cuBLAS_Time1<<std::endl;
    std::cout<<"CUBLAS Batched Strided Execution Time: "<<cuBLAS_Time2<<std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_batch_A);
    hipFree(d_batch_B);
    hipFree(d_batch_C);
    hipFree(d_A_array);
    hipFree(d_B_array);
    hipFree(d_C_array);
    
    hipblasDestroy(handle);
    
    return 0;
}