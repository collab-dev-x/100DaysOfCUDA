#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    const int M = 3, N = 5;
    const float alpha = 1.0f, beta = 0.5f;

    size_t matrix_A_size = M * N * sizeof(float);
    size_t matrix_B_size = M * N * sizeof(float);
    size_t matrix_C_size = M * M * sizeof(float);
    
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, matrix_A_size);
    hipMalloc(&d_B, matrix_B_size);
    hipMalloc(&d_C, matrix_C_size);
    
    std::vector<float> h_A_data(M * N, 1.0f);
    std::vector<float> h_B_data(M * N, 2.0f);
    std::vector<float> h_C_data(M * M, 0.0f);
    std::vector<float> h_C_result(M * M);
    
    hipMemcpy(d_A, h_A_data.data(), matrix_A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_data.data(), matrix_B_size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C_data.data(), matrix_C_size, hipMemcpyHostToDevice);

    float cuBLAS_Time1;

    hipEventRecord(start1);
    hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_B, M, &beta, d_C, M);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&cuBLAS_Time1, start1, stop1);
    hipMemcpy(h_C_result.data(), d_C, matrix_C_size, hipMemcpyDeviceToHost);
    
    std::cout<<"CUBLAS Symmetric Matrix Multiplication Execution Time: "<<cuBLAS_Time1<<std::endl;

    for(int i=0;i<M*M;i++){
        std::cout<<h_C_result[i]<<" ";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipblasDestroy(handle);
    
    return 0;
}